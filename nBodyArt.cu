#include "hip/hip_runtime.h"
//Optimized using shared memory and on chip memory 																																			
// nvcc nBodyArt.cu -o nBodyArt -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

FILE* ffmpeg;

#define PI 3.141592654
#define BLOCK 256

// Globals to be read in from parameter file.
int NumberOfBodies;
float TotalRunTime;
float Dt;
float G;
float H;
float Epsalon;
float MassOfBody;
float DiameterOfBody;
float* Diameter;
float VelocityMax;
float Drag;
int DrawRate;

// Other Globals
int Pause;
float4 *BodyPosition, *BodyVelocity, *BodyForce;
float4 *BodyPositionGPU, *BodyVelocityGPU, *BodyForceGPU;
float4 *BodyColor;
dim3 Blocks, Grids;
int DrawTimer;
float RunTime;
int* Buffer;
int MovieOn;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void readSimulationParameters();
void allocateMemory();
void setInitailConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void setup();

#include "./callBackFunctions.h"

void readSimulationParameters()
{
	ifstream data;
	string name;
	
	data.open("./simulationSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> NumberOfBodies;
		
		getline(data,name,'=');
		data >> TotalRunTime;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> G;
		
		getline(data,name,'=');
		data >> H;
		
		getline(data,name,'=');
		data >> Epsalon;
		
		getline(data,name,'=');
		data >> MassOfBody;
		
		getline(data,name,'=');
		data >> DiameterOfBody;
		
		getline(data,name,'=');
		data >> VelocityMax;
		
		getline(data,name,'=');
		data >> Drag;
		
		getline(data,name,'=');
		data >> DrawRate;
	}
	else
	{
		printf("\nTSU Error could not open simulationSetup file\n");
		exit(0);
	}
	data.close();
	
	printf("\n\n Parameter file has been read");
}

void allocateMemory()
{
	Blocks.x = BLOCK;
	Blocks.y = 1;
	Blocks.z = 1;
	
	Grids.x = (NumberOfBodies - 1)/Blocks.x + 1;
	Grids.y = 1;
	Grids.z = 1;
	
	BodyPosition = (float4*)malloc(NumberOfBodies*sizeof(float4));
	BodyVelocity = (float4*)malloc(NumberOfBodies*sizeof(float4));
	BodyForce    = (float4*)malloc(NumberOfBodies*sizeof(float4));
	BodyColor    = (float4*)malloc(NumberOfBodies*sizeof(float4));
	Diameter = (float*)malloc(NumberOfBodies*sizeof(float));
	
	hipMalloc( (void**)&BodyPositionGPU, NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyPositionGPU");
	hipMalloc( (void**)&BodyVelocityGPU, NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyDiameterOfBodyVelocityGPU");
	hipMalloc( (void**)&BodyForceGPU,    NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyForceGPU");
	
	
	printf("\n\n Memory has been allocated");
}

void setInitailConditions()
{
    float dx, dy, dz, d, d2;
    int test;
	time_t t;
	
	srand((unsigned) time(&t));
	for(int i = 0; i < NumberOfBodies; i++)
	{
		Diameter[i] = ((float)rand()/(float)RAND_MAX)*0.01 - 0.001;	
	}	
	for(int i = 0; i < NumberOfBodies; i++)
	{
		
		test = 0;
		while(test == 0)
		{
			// Get random number between -1 at 1.
			BodyPosition[i].x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			BodyPosition[i].y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			BodyPosition[i].z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			BodyPosition[i].w = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0; 	//MassOfBody;
			test = 1;
			
			for(int j = 0; j < i; j++)
			{
				dx = BodyPosition[i].x-BodyPosition[j].x;
				dy = BodyPosition[i].y-BodyPosition[j].y;
				dz = BodyPosition[i].z-BodyPosition[j].z;
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);
				
				if(d < DiameterOfBody)
				{
					test = 0;
					break;
				}
			}
			
			if(test == 1)
			{
				BodyVelocity[i].x = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				BodyVelocity[i].y = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				BodyVelocity[i].z = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				BodyVelocity[i].w = 0.0;
				
				BodyColor[i].x = ((float)rand()/(float)RAND_MAX);
				BodyColor[i].y = ((float)rand()/(float)RAND_MAX);
				BodyColor[i].z = ((float)rand()/(float)RAND_MAX);
				BodyColor[i].w = 0.0;
			}
		}
	}
	printf("\n\n Initail conditions have been set.");
}

void drawPicture()
{
	//glClear(GL_COLOR_BUFFER_BIT);
	//glClear(GL_DEPTH_BUFFER_BIT);
	
	for(int i = 0; i < NumberOfBodies; i++)
	{
		glColor3d(BodyColor[i].x, BodyColor[i].y, BodyColor[i].z);
		//glColor3d(1.0, 1.0, 1.0);
		glPushMatrix();
			glTranslatef(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
			glutSolidSphere(Diameter[i]/2.0, 30, 30);
			
			//glTranslatef(0.0, 0.0, 0.0);
			//glutSolidSphere(1.0, 20, 20);
		glPopMatrix();
	}
	glutSwapBuffers();
	
	
	if(MovieOn == 1)
	{
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ffmpeg);
	}
}
                                 
__device__ float3 getBodyBodyForce(float4 p0, float4 p1, float G, float H, float Epsalon)
{
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz + Epsalon;
    float r = sqrt(r2);
    
    float force  = (G*p0.w*p1.w)/(r2) - (H*p0.w*p1.w)/(r2*r2);
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

__global__ void getForces(float4 *pos, float4 *vel, float4 * force, float G, float H, float Epsalon, int n)
{
	int j,ii;
    float3 force_mag, forceSum;
    float4 posMe;
    __shared__ float4 shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	forceSum.z = 0.0;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	posMe.w = pos[id].w;
	    
    for(j=0; j < gridDim.x; j++)
    {
    	shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    	__syncthreads();
   
		#pragma unroll 32
        for(int i=0; i < blockDim.x; i++)	
        {
        	ii = i + blockDim.x*j;
		    if(ii != id && ii < n) 
		    {
		    	force_mag = getBodyBodyForce(posMe, shPos[i], G, H, Epsalon);
			    forceSum.x += force_mag.x;
			    forceSum.y += force_mag.y;
			    forceSum.z += force_mag.z;
		    }
	   	 }
	}
	if(id < n)
	{
	    force[id].x = forceSum.x;
	    force[id].y = forceSum.y;
	    force[id].z = forceSum.z;
    }
}

__global__ void moveBodies(float4 *pos, float4 *vel, float4 * force, float drag, float dt, int n)
{
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    if(id < n)
    {
	    vel[id].x += ((force[id].x-drag*vel[id].x)/pos[id].w)*dt;
	    vel[id].y += ((force[id].y-drag*vel[id].y)/pos[id].w)*dt;
	    vel[id].z += ((force[id].z-drag*vel[id].z)/pos[id].w)*dt;
	
	    pos[id].x += vel[id].x*dt;
	    pos[id].y += vel[id].y*dt;
	    pos[id].z += vel[id].z*dt;
	    
	    
    }
}

void nBody()
{
	//drawPicture();
	//while(1);
	if(Pause != 1)
	{	
		getForces<<<Grids, Blocks>>>(BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, G, H, Epsalon, NumberOfBodies);
		moveBodies<<<Grids, Blocks>>>(BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, Drag, Dt, NumberOfBodies);
        
        DrawTimer++;
		if(DrawTimer == DrawRate) 
		{
		    hipMemcpy( BodyPosition, BodyPositionGPU, NumberOfBodies*sizeof(float4), hipMemcpyDeviceToHost );
			drawPicture();
			//printf("\n Time = %f", RunTime);
			DrawTimer = 0;
		}
		RunTime += Dt; 
		if(TotalRunTime < RunTime)
		{
			printf("\n\n Done\n");
			exit(0);
		}
	}
}

void errorCheck(const char *message)
{
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess)
  {
    printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

void setup()
{	
	readSimulationParameters();
	allocateMemory();
	setInitailConditions();
	hipMemcpy( BodyPositionGPU, BodyPosition, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( BodyVelocityGPU, BodyVelocity, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
    DrawTimer = 0;
	RunTime = 0.0;
	Pause = 1;
}

int main(int argc, char** argv)
{
	setup();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 
	Buffer = new int[XWindowSize*YWindowSize];

	// Clip plains
	Near = 0.2;
	Far = 30.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 2.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("N Body");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mymouse);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutMainLoop();
	return 0;
}






